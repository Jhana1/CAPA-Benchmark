#include "hip/hip_runtime.h"
#include <cstdio>

template <typename T>
using uCat = T(*)(T);

template <typename T>
using mCat = T(*)(T, T);

__device__ int square(int a)
{
    return a * a;
}

__device__ int mult(int a, int b)
{
    return a * b;
}

template <typename T>
__device__ T mult(T a, T b)
{
    return a * b;
}

template <typename T>
__device__ T add2(int a, float b)
{
    return (T) (a + b);
}

template <typename T, mCat<T> func>
__global__ void biMapKernel(T *a, T *b, T *c, size_t size)
{
    size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size)
        c[i] = func(a[i], b[i]);
}

template <typename T, uCat<T> func>
__global__ void MapKernel(T *a, T *c, size_t size)
{
    size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size)
        c[i] = func(a[i]);
}

template <typename T, mCat<T> func>
__global__ void ReduceKernel(const T *a, T *c, size_t size)
{
    extern __shared__ T sdata[];

    size_t myId = threadIdx.x + blockIdx.x * blockDim.x;
    size_t tid  = threadIdx.x;

    if (myId > size)
        return;

    sdata[tid] = a[myId];
    __syncthreads();

    for (size_t i = blockDim.x/2; i > 0; i /= 2)
    {
        if (tid < i)
        {
            sdata[tid] = func(sdata[tid], sdata[tid+i]);
        }
        __syncthreads();
    }
    if (tid == 0)
    {
        c[blockIdx.x] = sdata[0];
    }
}

template <typename T>
void arrPrint(T *a, size_t l)
{
    for (size_t i = 0; i < l; ++i)
    {
        printf("%d ", a[i]);
    }
    printf("\n");
}
int main()
{
    int h_a[10], h_b[10], h_c[10];
    int *d_a, *d_b, *d_c;

    for (int i = 0; i < 10; ++i)
    {
        h_a[i] = h_b[9-i] = -i;
    }

    arrPrint(h_a,10);
    arrPrint(h_b,10);
    arrPrint(h_c,10);

    hipMalloc((void **) &d_a, 10*sizeof(int));
    hipMalloc((void **) &d_b, 10*sizeof(int));
    hipMalloc((void **) &d_c, 10*sizeof(int));
    hipMemcpy(d_a, h_a, 10 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, 10 * sizeof(int), hipMemcpyHostToDevice);

    biMapKernel<int,mult><<<10,1>>>(d_a, d_b, d_c, 10);

    hipMemcpy(h_c, d_c, 10 * sizeof(int), hipMemcpyDeviceToHost);

    arrPrint(h_c,10);
    return 0;
}
