#include "benchmarkObj.h"
#include "benchmarkKernels.cuh"
#include "miscFunctions.h"

BenchmarkObj::BenchmarkObj()
{
    mDevice = 0;
    hipGetDeviceProperties(&mDeviceInfo, mDevice);
}

BenchmarkObj::BenchmarkObj(int device)
{
    mDevice = device;
    hipGetDeviceProperties(&mDeviceInfo, mDevice);
}

std::string BenchmarkObj::DeviceName()
{
    return std::string(mDeviceInfo.name);
}

int BenchmarkObj::TheoMemBandwidth()
{
    int memoryClockRate = mDeviceInfo.memoryClockRate;
    int memoryBusWidth = mDeviceInfo.memoryBusWidth;
    return memoryClockRate * (memoryBusWidth/8);
}

void BenchmarkObj::BenchMemBandwidth()
{
    size_t size = 10;
    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;
    
    h_a = (int*) malloc(sizeof(int) * size);
    h_b = (int*) malloc(sizeof(int) * size);
    h_c = (int*) malloc(sizeof(int) * size);

    vectorFill(h_a, size);
    vectorFill(h_b, size);
    vectorPrint(h_a, size);
    vectorPrint(h_b, size);

    hipMalloc((void **) &d_a, sizeof(int) * size);
    hipMalloc((void **) &d_b, sizeof(int) * size);
    hipMalloc((void **) &d_c, sizeof(int) * size);

    hipMemcpy(d_a, h_a, sizeof(int) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int) * size, hipMemcpyHostToDevice);

    addKernel<<<1024,1024>>>(d_a, d_b, d_c, size);
    addKernelT<int><<<1024,1024>>>(d_c, d_c, d_c, size);

    hipMemcpy(h_c, d_c, sizeof(int) * size, hipMemcpyDeviceToHost);
    vectorPrint(h_c, size);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);


}
