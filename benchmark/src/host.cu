#include "hip/hip_runtime.h"
// Project Headers
#include "functors.h"
#include "bench.h"
#include "benchFixture.h"

// Third Party Headers
#include "hayai.hpp"

// STL Headers
//#include <cstdlib>
#include <iostream>

typedef BenchFixture<float>  FloatFixture;
typedef BenchFixture<double> DoubleFixture;
typedef BenchFixture<int>    IntFixture;

volatile double acc = 0;


BENCHMARK_F(FloatFixture, HostMinReduce, 100, 10)
{
    AAA::min<float> binaryOp;
    acc += bench->host_reduce(binaryOp);
}

BENCHMARK_F(FloatFixture, DeviceMinReduceWithOnload, 100, 10)
{
    AAA::min<float> binaryOp;
    bench->device_reduce_onload(binaryOp);
}

BENCHMARK_F(FloatFixture, HostMatrixMult, 10, 1)
{
    bench->host_matrix_mult();
}

BENCHMARK_F(FloatFixture, DeviceMatrixMult, 10, 1)
{
    bench->device_matrix_mult();
}

int main()
{
    hayai::ConsoleOutputter consoleOutputter;//(std::cout);
    hayai::Benchmarker::AddOutputter(consoleOutputter);
    
    Bench<float> bb;
    AAA::max<float> binaryOp;

    bb.prepare();
    bb.cuBLAS_prepare();
    hayai::Benchmarker::RunAllTests();
    bb.cuBLAS_destroy();
    return 0;
}
