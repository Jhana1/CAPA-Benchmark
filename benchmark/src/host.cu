#include "hip/hip_runtime.h"
// Project Headers
#include "functors.h"
#include "bench.h"
#include "benchInfo.h"
#include "benchFixture.h"

// Third Party Headers
#include "hayai.hpp"

// STL Headers
//#include <cstdlib>
#include <iostream>

// Benchmark Includes
// Must include typedefs first
#include "benchmarkTypedefs.h"
#include "reductionBenchmarks.h"
#include "matrixBenchmarks.h"

int main(int argc, char *argv[])
{
    if (argc > 1)
    {
        hayai::JsonOutputter consoleOutputter(std::cout);
        hayai::Benchmarker::AddOutputter(consoleOutputter);
    }
    else
    {
        hayai::ConsoleOutputter consoleOutputter;
        hayai::Benchmarker::AddOutputter(consoleOutputter);
    }
    
    Bench<float, case6> bb;
    AAA::max<float> binaryOp;

    bb.prepare();
    bb.cuBLAS_prepare();
    hayai::Benchmarker::RunAllTests();
    bb.cuBLAS_destroy();
    return 0;
}
