#include "hip/hip_runtime.h"
#include "benchmarkKernels.cuh"

__global__
void addKernel(int *vecA, int *vecB, int *vecC, size_t size)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < size)
        vecC[id] = vecA[id] + vecB[id];
}
